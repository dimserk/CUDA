#include <stdio.h>
#include <ctime>
#include "hip/hip_runtime.h"

#define array_len 200

__global__ void add(int *a, int *b, int *c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	
	int size = sizeof(int) * array_len;
	
	a = (int*)malloc(size);
	b = new int[array_len];
	c = new int[array_len];
	
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	std::srand(std::time(0));
	for(int i = 0; i < array_len; i++)
	{
		a[i] = 1 + std::rand() % 100;
		b[i] = 1 + std::rand() % 100;
	}
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	add<<<array_len,1>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("Calculated\n");
	for(int i = 0; i < array_len; i++)
		printf("%-3d + %-3d = %-3d\n", a[i], b[i], c[i]);
		
	free(a);
	free(b);
	free(c);

	return 0;
}