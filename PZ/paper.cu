#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

__global__ void action(int *array1, int* array2, int* array_res) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    switch(i % 3)
    {
    case 0:
        array_res[i] = array1[i] + 1;
        break;
    case 1:
        array_res[i] = array2[i] - 1;
        break;
    case 2:
        array_res[i] = array1[i] * array2[i];
        break;
    }
}

//////////////////
struct is_first
{
    int p = -1;
    __device__ __host__ bool operator() (int x)
    {
        return x ;
    }
};

template<typename T>
struct first_func
{
    __device__ __host__ T operator ()( const T& x ) const
    {
        return x + 1;
    }
};

template<typename T>
struct second_func
{
    __device__ __host__ T operator ()( const T& x ) const
    {
        return x - 1;
    }
};

template<typename T>
struct third_func
{
    __device__ __host__ T operator ()( const T& x ) const
    {
        return x + 1;
    }
};
//////////////////

void array_print(int* array, int array_len, const char* message)
{
    cout << " " << message << ":\n [ ";
    for (int i = 0; i < array_len; i++)
    {
        cout << array[i] << " ";
    }
    cout << "]" << endl;
}

int main(int argc, char** argv)
{
    int array_len, start, stop;
    
    //Obtaining command line arguments
    switch (argc)
    {
    case 1:
        array_len = 15;
        cout << " #Warning# Default array size: " << array_len << endl;
        start = 0;
        cout << " #Warning# Default random start: " << start << endl;
        stop = 100;
        cout << " #Warning# Default random stop: " << stop << endl;
        cout << endl;
        break;
    case 2:
        array_len = atoi(argv[1]);
        start = 0;
        cout << " #Warning# Default random start: " << start << endl;
        stop = 100;
        cout << " #Warning# Default random stop: " << stop << endl;
        cout << endl;
        break;
    case 4:
        array_len = atoi(argv[1]);
        start = atoi(argv[2]);
        stop = atoi(argv[3]);
        cout << endl;
        break;   
    default:
        cout << "Wrong input!" << endl;
    }

    //Prepairing variables
    int *array1 = new int[array_len];
    int *array2 = new int[array_len];
    int *array_res =  new int[array_len];
    int * d_array1, *d_array2, *d_array_res;

    // thrust::device_vector<int> t_array1(array_len);
    // thrust::device_vector<int> t_array2(array_len);
    // thrust::device_vector<int> t_array_res(array_len);

    hipMalloc((void**)&d_array1, sizeof(int)*array_len);
    hipMalloc((void**)&d_array2, sizeof(int)*array_len);
    hipMalloc((void**)&d_array_res, sizeof(int)*array_len);

    //Randomizing array
    srand(time(NULL));
    for (int i = 0; i < array_len; i++)
    {
        array1[i] = start + rand() % stop;
        //t_array1[i] = array1[i];
        array2[i] = start + rand() % stop;
        //t_array2[i] = array2[i];
    }

    array_print(array1, array_len, "Array1");
    array_print(array2, array_len, "Array2");

    //Some copies 
    hipMemcpy(d_array1, array1, sizeof(int) * array_len, hipMemcpyHostToDevice);
    hipMemcpy(d_array2, array2, sizeof(int) * array_len, hipMemcpyHostToDevice);

    action<<<array_len, 1>>>(d_array1, d_array2, d_array_res);
    hipDeviceSynchronize();

    //thrust::transform_if(thrust::device, t_array1.begin(), t_array1.end(),
    //                     t_array2.begin(), first_func<int>(), is_first());
    // for(int i = 0; i < array_len; i++)
    // {
    //     cout << t_array2[i] << " ";
    // }
    // cout << endl;

    //Some copies 
    hipMemcpy(array_res, d_array_res, sizeof(int) * array_len, hipMemcpyDeviceToHost);
    
    array_print(array_res, array_len, "Res array");
    
    delete[] array1, array2, array_res;

    hipFree(d_array1);
    hipFree(d_array2);
    hipFree(d_array_res);

    hipDeviceReset();

    return 0;
}