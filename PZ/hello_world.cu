
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{
	printf("Hello world from device!\n");
}

int main()
{
	printf("Hello world from host!\n");
	
	hello<<<10,1>>>();
	hipDeviceSynchronize();
	
	return 0;
}
	