#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

__global__ void kernel(int *num)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("ID:%d - bIdx:%d, bDim:%d, tIdx:%d\n", id, blockIdx.x, blockDim.x, threadIdx.x);
    (*num)++;
}

int main(int argc, char** argv)
{
    int num = 0;
    int *d_num;

    hipMalloc((void**)&d_num, sizeof(int));
    hipMemcpy(d_num, &num, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<2, 3>>>();
    hipDeviceSynchronize();

    hipMemcpy(num, d_num, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_num);

    return 0;
}
