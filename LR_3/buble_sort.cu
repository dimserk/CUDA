#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include "hip/hip_runtime.h"

#define PRTINTING_LIMIT 20

using namespace std;

__global__ void g_buble_sort(int *array, int* array_len, int* iter_num)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    bool work = false;
    int position = 0;

    while(true)
    {
        if(i * 2 == *iter_num)
        {
            work = true;
        }

        if (work)
        {
            if(array[position] > array[position  + 1])
            {
                int tmp = array[position];
                array[position] = array[position + 1];
                array[position + 1] = tmp;
            }

            position++;
        }

        if(i == 0)
        {
            (*iter_num)++;
        }

        if(position == *array_len - 1)
        {
            work = false;
            if(i != 0)
            {
                break;
            }
        }

        if(*iter_num == *array_len * 2 - 3)
        {
            break;
        }

        __syncthreads();
    }
}

void buble_sort(int* array, int array_len)
{
    for(int i = 1; i < array_len - 1; i++)
    {
        for(int j = 0; j < array_len - i; j++)
        {
            if(array[j] > array[j + 1])
            {  
                int tmp = array[j];
                array[j] = array[j + 1];
                array[j + 1] = tmp;
            }
        }
    }
}

void array_print(int* array, int array_len, const char* message)
{
    cout << " " << message << ":\n [ ";
    for (int i = 0; i < array_len; i++)
    {
        cout << array[i] << " ";
    }
    cout << "]" << endl;
}

int main(int argc, char** argv)
{
    int array_len = 15, start = 0, stop = 100;
    
    //Obtaining command line arguments
    switch (argc)
    {
    case 1:
        cout << " #Warning# Default array size: " << array_len << endl;
        cout << " #Warning# Default random start: " << start << endl;
        cout << " #Warning# Default random stop: " << stop << endl;
        break;
    case 2:
        array_len = atoi(argv[1]);
        cout << " #Warning# Default random start: " << start << endl;
        cout << " #Warning# Default random stop: " << stop << endl;
        break;
    case 4:
        array_len = atoi(argv[1]);
        start = atoi(argv[2]);
        stop = atoi(argv[3]);
        break;   
    default:
        cout << " #Error# Wrong input! Default settings applied." << endl;
        cout << " #Warning# Default array size: " << array_len << endl;
        cout << " #Warning# Default random start: " << start << endl;
        cout << " #Warning# Default random stop: " << stop << endl;
    }
    cout << endl;

    if(array_len < 2)
    {
        cout << " #Error# Array length is too small, at least 2!" << endl;
        return 0;
    }

    //Prepairing variables
    int *init_array = new int[array_len], *gpu_array = new int[array_len];
    int *d_array, *d_array_len, *d_iter_num;
    int array_size = sizeof(int) * array_len;
    float working_time = 0;

    hipEvent_t e_start, e_stop;

    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    hipMalloc((void**)&d_array, array_size);
    hipMalloc((void**)&d_array_len, sizeof(int));
    hipMalloc((void**)&d_iter_num, sizeof(int));

    //Randomizing array
    srand(time(NULL));
    for (int i = 0; i < array_len; i++)
    {
        init_array[i] = start + rand() % stop;
    }

    //Copy array
    memcpy(gpu_array, init_array, array_size);
    hipMemcpy(d_array, gpu_array, array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_array_len, &array_len, sizeof(int), hipMemcpyHostToDevice);
    int iter_num = 0;
    hipMemcpy(d_iter_num, &iter_num, sizeof(int), hipMemcpyHostToDevice);

    if(array_len <= PRTINTING_LIMIT)
    {
        array_print(init_array, array_len, "Initial array");
    }

    //GPU sorting
    hipEventRecord(e_start);

    g_buble_sort<<<array_len - 1, 1>>>(d_array, d_array_len, d_iter_num);
    hipError_t cuda_status = hipGetLastError();
    if(cuda_status != hipSuccess)
    {
        cout << "Kernel error!" << endl;
        goto cuda_error;
    }

    hipDeviceSynchronize();
    
    hipEventRecord(e_stop);
    hipEventSynchronize(e_stop);
    hipEventElapsedTime(&working_time, e_start, e_stop);

    hipMemcpy(gpu_array, d_array, array_size, hipMemcpyDeviceToHost);
    
    //GPU printing
    if(array_len <= PRTINTING_LIMIT)
    {
        array_print(gpu_array, array_len, "Array after GPU sort");
    }
    cout << " GPU sorting time: " << working_time << " ms" << endl;

    //CPU sorting
    clock_t c_start = clock();
    buble_sort(init_array, array_len);
    clock_t c_end = clock();

    //CPU printing
    if(array_len <= PRTINTING_LIMIT)
    {
        array_print(init_array, array_len, "Array after CPU sort");
    }
    cout << " CPU sorting time: " << 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC << " ms" << endl;

cuda_error:
    delete[] init_array, gpu_array;

    hipEventDestroy(e_start);
    hipEventDestroy(e_stop);

    hipFree(d_array);
    hipFree(d_array_len);
    hipFree(d_iter_num);

    return 0;
}
