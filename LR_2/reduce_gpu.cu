#include <iostream>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include "hip/hip_runtime.h"

using namespace std;

__global__ void kernel(int *array, int *i)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int second =  (1 << *i + 1)  + (1 << *i + 1) * j - 1;
    int first = second - (1 << *i);

    array[second] = array[first] + array[second];
}

int main(int argc, char** argv)
{
    int real_array_size = 512, start = 0, stop = 100;
    
    //Obtaining command line arguments
    switch (argc)
    {
    case 1:
        cout << " #Warning# Default array size: " << real_array_size << endl;
        cout << " #Warning# Default random start: " << start << endl;
        cout << " #Warning# Default random stop: " << stop << endl;
        break;
    case 2:
        real_array_size = atoi(argv[1]);
        cout << " #Warning# Default random start: " << start << endl;
        cout << " #Warning# Default random stop: " << stop << endl;
        break;
    case 4:
        real_array_size = atoi(argv[1]);
        start = atoi(argv[2]);
        stop = atoi(argv[3]);
        break;   
    default:
        cout << " #Error# Wrong input! Default settings applied." << endl;
        cout << " #Warning# Default array size: " << real_array_size << endl;
        cout << " #Warning# Default random start: " << start << endl;
        cout << " #Warning# Default random stop: " << stop << endl;
    }
    cout << endl;

    if(real_array_size < 2)
    {
        cout << " #Error# Array size is too small, at least 2!" << endl;
        return 0 ;
    }

    //Every array size can be used!
    int tmp_size, array_size, degree = 1;
    while(true)
    {
        tmp_size = pow(2, degree);
        if(real_array_size <= tmp_size)
        {
            array_size = tmp_size;
            break;
        } 

        degree++;
    }
    int *array = new int[array_size];

    //Randomazing array
    srand(time(NULL));
    for(int i = 0; i < array_size; i++)
    {
        if(i <= real_array_size - 1)
        {
            array[i] = start + rand() % stop;
        }
        else
        {
            array[i] = 0;
        }
    }

    //Control summation
    int cpu_sum = 0;
    for(int i = 0; i < real_array_size; i++)
    {
        cpu_sum += array[i];
    }

    //Device varaibles
    int *d_array, *d_i;
    int size = sizeof(int) * array_size;
    float working_time = 0;

    hipEvent_t e_start, e_stop;

    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    hipMalloc((void**)&d_array, size);
    hipMalloc((void**)&d_i, sizeof(int));

    hipMemcpy(d_array, array, size, hipMemcpyHostToDevice);

    hipEventRecord(e_start);

    int iteration_num = array_size;
    for (int i = 0; i < log10(array_size)/log10(2); i++)
    {
        iteration_num /= 2;

        hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);

        kernel<<<iteration_num, 1>>>(d_array, d_i);
        hipDeviceSynchronize();

        hipError_t cuda_status = hipGetLastError();
        if(cuda_status != hipSuccess)
        {
            cout << " #Error# Kernel error!" << endl;
            goto cuda_error;
        }
    }

    hipEventRecord(e_stop);

    hipMemcpy(array, d_array, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(e_stop);
    hipEventElapsedTime(&working_time, e_start, e_stop);

    //Printing result
    cout << " GPU summation time: " << working_time << " ms" << endl;
    cout << " Total sum of the array: " << array[array_size - 1] << " (GPU)" << endl;
    cout << " Total sum of the array: " << cpu_sum << " (CPU)" << endl;

cuda_error:
    delete[] array;

    hipFree(d_array);
    hipFree(d_i);

    hipEventDestroy(e_start);
    hipEventDestroy(e_stop);

    return 0;
}
